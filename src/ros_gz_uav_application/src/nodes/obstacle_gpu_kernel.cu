#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

// 障碍物数据结构（GPU版本）
struct GPUMarker {
    int x, y, z;
    unsigned char cost;
};

// CUDA核函数：标记长方体障碍物
__global__ void markBoxObstacleKernel(
    unsigned char* voxel_grid,
    int grid_x, int grid_y, int grid_z,
    double resolution_xy, double resolution_z,
    double center_x, double center_y, double center_z,
    double size_x, double size_y, double size_z,
    double buffer_size,
    unsigned char cost_value
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= grid_x || y >= grid_y || z >= grid_z) return;
    
    // 计算世界坐标
    double world_x = x * resolution_xy;
    double world_y = y * resolution_xy;
    double world_z = z * resolution_z;
    
    // 计算障碍物边界
    double min_x = center_x - size_x/2 - buffer_size;
    double max_x = center_x + size_x/2 + buffer_size;
    double min_y = center_y - size_y/2 - buffer_size;
    double max_y = center_y + size_y/2 + buffer_size;
    double min_z = center_z - size_z/2 - buffer_size;
    double max_z = center_z + size_z/2 + buffer_size;
    
    // 检查是否在障碍物内
    if (world_x >= min_x && world_x <= max_x &&
        world_y >= min_y && world_y <= max_y &&
        world_z >= min_z && world_z <= max_z) {
        
        int index = z * grid_x * grid_y + y * grid_x + x;
        voxel_grid[index] = cost_value;
    }
}

// CUDA核函数：标记圆柱体障碍物
__global__ void markCylinderObstacleKernel(
    unsigned char* voxel_grid,
    int grid_x, int grid_y, int grid_z,
    double resolution_xy, double resolution_z,
    double center_x, double center_y, double center_z,
    double radius, double height,
    double buffer_size,
    unsigned char cost_value
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= grid_x || y >= grid_y || z >= grid_z) return;
    
    // 计算世界坐标
    double world_x = x * resolution_xy;
    double world_y = y * resolution_xy;
    double world_z = z * resolution_z;
    
    // 计算XY平面距离
    double dx = world_x - center_x;
    double dy = world_y - center_y;
    double dist_sq = dx*dx + dy*dy;
    double radius_sq = (radius + buffer_size) * (radius + buffer_size);
    
    // 检查Z轴范围
    double min_z = center_z - height/2 - buffer_size;
    double max_z = center_z + height/2 + buffer_size;
    
    // 检查是否在圆柱内
    if (dist_sq <= radius_sq && world_z >= min_z && world_z <= max_z) {
        int index = z * grid_x * grid_y + y * grid_x + x;
        voxel_grid[index] = cost_value;
    }
}

// CUDA核函数：标记球体障碍物
__global__ void markSphereObstacleKernel(
    unsigned char* voxel_grid,
    int grid_x, int grid_y, int grid_z,
    double resolution_xy, double resolution_z,
    double center_x, double center_y, double center_z,
    double radius,
    double buffer_size,
    unsigned char cost_value
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= grid_x || y >= grid_y || z >= grid_z) return;
    
    // 计算世界坐标
    double world_x = x * resolution_xy;
    double world_y = y * resolution_xy;
    double world_z = z * resolution_z;
    
    // 计算3D距离
    double dx = world_x - center_x;
    double dy = world_y - center_y;
    double dz = world_z - center_z;
    double dist_sq = dx*dx + dy*dy + dz*dz;
    double radius_sq = (radius + buffer_size) * (radius + buffer_size);
    
    // 检查是否在球体内
    if (dist_sq <= radius_sq) {
        int index = z * grid_x * grid_y + y * grid_x + x;
        voxel_grid[index] = cost_value;
    }
}

// 主机端包装函数
extern "C" {
    // 标记长方体障碍物
    int markBoxObstacleGPU(
        unsigned char* voxel_grid,
        int grid_x, int grid_y, int grid_z,
        double resolution_xy, double resolution_z,
        double center_x, double center_y, double center_z,
        double size_x, double size_y, double size_z,
        double buffer_size,
        unsigned char cost_value
    ) {
        // 分配GPU内存
        unsigned char* d_voxel_grid;
        size_t grid_size = grid_x * grid_y * grid_z * sizeof(unsigned char);
        hipMalloc(&d_voxel_grid, grid_size);
        
        // 复制数据到GPU
        hipMemcpy(d_voxel_grid, voxel_grid, grid_size, hipMemcpyHostToDevice);
        
        // 设置CUDA网格和块大小
        dim3 block_size(16, 16, 4);
        dim3 grid_size_cuda(
            (grid_x + block_size.x - 1) / block_size.x,
            (grid_y + block_size.y - 1) / block_size.y,
            (grid_z + block_size.z - 1) / block_size.z
        );
        
        // 启动核函数
        markBoxObstacleKernel<<<grid_size_cuda, block_size>>>(
            d_voxel_grid, grid_x, grid_y, grid_z,
            resolution_xy, resolution_z,
            center_x, center_y, center_z,
            size_x, size_y, size_z,
            buffer_size, cost_value
        );
        
        // 复制结果回主机
        hipMemcpy(voxel_grid, d_voxel_grid, grid_size, hipMemcpyDeviceToHost);
        
        // 清理GPU内存
        hipFree(d_voxel_grid);
        
        return 0;
    }
    
    // 标记圆柱体障碍物
    int markCylinderObstacleGPU(
        unsigned char* voxel_grid,
        int grid_x, int grid_y, int grid_z,
        double resolution_xy, double resolution_z,
        double center_x, double center_y, double center_z,
        double radius, double height,
        double buffer_size,
        unsigned char cost_value
    ) {
        // 分配GPU内存
        unsigned char* d_voxel_grid;
        size_t grid_size = grid_x * grid_y * grid_z * sizeof(unsigned char);
        hipMalloc(&d_voxel_grid, grid_size);
        
        // 复制数据到GPU
        hipMemcpy(d_voxel_grid, voxel_grid, grid_size, hipMemcpyHostToDevice);
        
        // 设置CUDA网格和块大小
        dim3 block_size(16, 16, 4);
        dim3 grid_size_cuda(
            (grid_x + block_size.x - 1) / block_size.x,
            (grid_y + block_size.y - 1) / block_size.y,
            (grid_z + block_size.z - 1) / block_size.z
        );
        
        // 启动核函数
        markCylinderObstacleKernel<<<grid_size_cuda, block_size>>>(
            d_voxel_grid, grid_x, grid_y, grid_z,
            resolution_xy, resolution_z,
            center_x, center_y, center_z,
            radius, height,
            buffer_size, cost_value
        );
        
        // 复制结果回主机
        hipMemcpy(voxel_grid, d_voxel_grid, grid_size, hipMemcpyDeviceToHost);
        
        // 清理GPU内存
        hipFree(d_voxel_grid);
        
        return 0;
    }
    
    // 标记球体障碍物
    int markSphereObstacleGPU(
        unsigned char* voxel_grid,
        int grid_x, int grid_y, int grid_z,
        double resolution_xy, double resolution_z,
        double center_x, double center_y, double center_z,
        double radius,
        double buffer_size,
        unsigned char cost_value
    ) {
        // 分配GPU内存
        unsigned char* d_voxel_grid;
        size_t grid_size = grid_x * grid_y * grid_z * sizeof(unsigned char);
        hipMalloc(&d_voxel_grid, grid_size);
        
        // 复制数据到GPU
        hipMemcpy(d_voxel_grid, voxel_grid, grid_size, hipMemcpyHostToDevice);
        
        // 设置CUDA网格和块大小
        dim3 block_size(16, 16, 4);
        dim3 grid_size_cuda(
            (grid_x + block_size.x - 1) / block_size.x,
            (grid_y + block_size.y - 1) / block_size.y,
            (grid_z + block_size.z - 1) / block_size.z
        );
        
        // 启动核函数
        markSphereObstacleKernel<<<grid_size_cuda, block_size>>>(
            d_voxel_grid, grid_x, grid_y, grid_z,
            resolution_xy, resolution_z,
            center_x, center_y, center_z,
            radius,
            buffer_size, cost_value
        );
        
        // 复制结果回主机
        hipMemcpy(voxel_grid, d_voxel_grid, grid_size, hipMemcpyDeviceToHost);
        
        // 清理GPU内存
        hipFree(d_voxel_grid);
        
        return 0;
    }
} 